#include "hip/hip_runtime.h"
// Added by Karel Adamek 

#ifndef BLN_KERNEL_H_
#define BLN_KERNEL_H_

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "AstroAccelerate/params.h"

__global__ void BLN_MSD_GPU_grid(float const* __restrict__ d_input, float *d_output, int x_steps, int y_steps, int nColumns, int msd) {
	extern __shared__ float Ms_Ss[];
	
	int warp_id, local_id, dim_y, pos;
	float x; // current element
	float M; // streaming mean
	float S; // streaming sum of squares (x_i-\bar{x})
	float j;
	float ftemp;
	
	local_id = threadIdx.x & (WARP - 1);
	warp_id = threadIdx.x>>5;
	dim_y = blockDim.x>>5;
	
	//----------------------------------------------
	//---- Calculating of streaming mean and sum of squares
	pos = (blockIdx.y*dim_y + warp_id)*y_steps*nColumns + blockIdx.x*WARP*x_steps + local_id;
	M=__ldg(&d_input[pos]);
	S=0;
	j=1.0f;
	for(int xf=1; xf<x_steps; xf++){
		pos = pos + WARP;
		x = __ldg(&d_input[pos]);
		j = j+1.0f;
		M = M + x;
		ftemp = (j*x - M);
		S = S + 1.0f/(j*(j-1.0f))*ftemp*ftemp;			
	}
	
	pos = pos + nColumns - (x_steps-1)*WARP;
	for(int yf=1; yf<y_steps; yf++){
		for(int xf=0; xf<x_steps; xf++){
			x = __ldg(&d_input[pos]);
			j = j+1.0f;
			M = M + x;
			ftemp = (j*x - M);
			S = S + 1.0f/(j*(j-1.0f))*ftemp*ftemp;
			pos = pos + WARP;
		}
		pos = pos + nColumns - x_steps*WARP;
	}
	
	Ms_Ss[threadIdx.x]=M;
	Ms_Ss[blockDim.x + threadIdx.x]=S;
	
	__syncthreads();
	
	// now all threads had saved their work, reduction follows
	
	// first we must load initial values
	//j=Neco;
	for(int i=(blockDim.x>>1); i>HALF_WARP; i=i>>1){
		if(threadIdx.x<i){
			j=j*2;
			ftemp = (M - Ms_Ss[i + threadIdx.x]);
			S = S + Ms_Ss[blockDim.x + i + threadIdx.x] + (1.0f/j)*ftemp*ftemp;
			M = M + Ms_Ss[i + threadIdx.x];
			
			Ms_Ss[threadIdx.x]=M;
			Ms_Ss[blockDim.x + threadIdx.x]=S;
		}
		// in the last iteration we do not need to save the results... or synchronize threads...
		__syncthreads();
	}
	
	// by now we should have only 32 partial results. shuffle reduction follows
	for(int q=HALF_WARP; q>0; q=q>>1){
		j=j*2;
		ftemp = (M - __shfl_down(M, q));
		S = S + __shfl_down(S, q) + (1.0f/j)*ftemp*ftemp;
		M = M + __shfl_down(M, q);
	}
	
	//----------------------------------------------
	//---- Writing data
	if(threadIdx.x==0){
		pos = blockIdx.y*gridDim.x + blockIdx.x;
		if(msd) {
			// produce mean and sd instead of T and S
			d_output[3*pos] = M/j;
			d_output[3*pos + 1] = sqrt(S/j);
		}
		else {
			d_output[3*pos] = M;
			d_output[3*pos + 1] = S;
		}
	}
}



__global__ void BLN_outlier_rejection(float const* __restrict__ d_input, float *d_output, float *d_stats, int size, float nElements, int nIterations, float multiplier) {
	__shared__ float Ms[WARP*WARP];
	__shared__ float Ss[WARP*WARP];
	__shared__ float js[WARP*WARP];
	__shared__ float s_signal_mean;
	__shared__ float s_signal_sd;
	
	
	int warp_id, pos;
	float M, Mt, S, j, jv;
	float ftemp;
	float signal_mean, signal_sd;
	
	__shared__ float s_ss_xx[WARP*WARP];
	__shared__ float s_ss_x[WARP*WARP];
	float ss_temp;
	float ss_xx;
	float ss_x;
	
	warp_id = threadIdx.x>>5;
	
	//-----------------------------------------------------------------
	//---- Calculation of the initial MSD
	pos=threadIdx.x;
	if(size>blockDim.x){
		M=__ldg(&d_input[3*pos]);
		S=__ldg(&d_input[3*pos+1]);
		j=nElements;
		//-------------------------- STATS --------------------------
		ss_temp = sqrt(__ldg(&d_input[3*pos+1])/((float) nElements));
		ss_x    = ss_temp;
		ss_xx   = (ss_temp*ss_temp);
		//-------------------------- STATS --------------------------
		pos = pos + blockDim.x;
		while (pos<size){
			jv=nElements;
			ftemp = ( jv/j*M - __ldg(&d_input[3*pos]) );
			S = S + __ldg(&d_input[3*pos+1]) + (j/(jv*(j+jv)))*ftemp*ftemp;
			M = M + __ldg(&d_input[3*pos]);
			j=j+jv;
			pos = pos + blockDim.x;
			//-------------------------- STATS --------------------------
			ss_temp = sqrt(__ldg(&d_input[3*pos+1])/((float) nElements));
			ss_x  += ss_temp;
			ss_xx += (ss_temp*ss_temp);
			//-------------------------- STATS --------------------------
		}
		
		__syncthreads();
		
		Ms[threadIdx.x]=M;
		Ss[threadIdx.x]=S;
		js[threadIdx.x]=j;
		
		//-------------------------- STATS --------------------------
		s_ss_x[threadIdx.x]  = ss_x;
		s_ss_xx[threadIdx.x] = ss_xx;
		//-------------------------- STATS --------------------------
		
		// now all threads had saved their work, reduction follows		
		// first we must load initial values
		for(int i=(blockDim.x>>1); i>HALF_WARP; i=i>>1){
			if(threadIdx.x<i){
				jv=js[i + threadIdx.x];
				ftemp = (jv/j*M - Ms[i + threadIdx.x]);
				S = S + Ss[i + threadIdx.x] + (j/(jv*(j+jv)))*ftemp*ftemp;
				M = M + Ms[i + threadIdx.x];
				j=j+jv;
				
				//-------------------------- STATS --------------------------
				ss_x  = ss_x + s_ss_x[threadIdx.x + i];
				ss_xx = ss_xx + s_ss_xx[threadIdx.x + i];
				//-------------------------- STATS --------------------------
				
				Ms[threadIdx.x]=M;
				Ss[threadIdx.x]=S;
				js[threadIdx.x]=j;
				//-------------------------- STATS --------------------------
				s_ss_x[threadIdx.x]  = ss_x;
				s_ss_xx[threadIdx.x] = ss_xx;
				//-------------------------- STATS --------------------------
			}
			__syncthreads();
		}
		
		// by now we should have only 32 partial results. shuffle reduction follows
		for(int q=HALF_WARP; q>0; q=q>>1){
			jv=__shfl_down(j, q);
			ftemp = (jv/j*M - __shfl_down(M, q));
			S = S + __shfl_down(S, q) + (j/(jv*(j+jv)))*ftemp*ftemp;
			M = M + __shfl_down(M, q);
			j=j+jv;
			
			//-------------------------- STATS --------------------------
			ss_x  = ss_x + __shfl_down(ss_x, q);
			ss_xx = ss_xx + __shfl_down(ss_xx, q);
			//-------------------------- STATS --------------------------
		}
		
	}
	else {
		if(threadIdx.x==0){
			//printf("Doing one thread thing\n");
			pos=0;
			M=__ldg(&d_input[3*pos]);
			S=__ldg(&d_input[3*pos+1]);
			j=nElements;
			//-------------------------- STATS --------------------------
			ss_temp = sqrt(__ldg(&d_input[3*pos+1])/((float) nElements));
			ss_x    = ss_temp;
			ss_xx   = (ss_temp*ss_temp);
			//-------------------------- STATS --------------------------
			for(pos=1; pos<size; pos++){
				jv=__ldg(&d_input[3*pos+2]);
				ftemp = ( jv/j*M - __ldg(&d_input[3*pos]) );
				S = S + __ldg(&d_input[3*pos+1]) + (j/(jv*(j+jv)))*ftemp*ftemp;
				M = M + __ldg(&d_input[3*pos]);
				j=j+jv;
				
				//-------------------------- STATS --------------------------
				ss_temp = sqrt(__ldg(&d_input[3*pos+1])/((float) nElements));
				ss_x  += ss_temp;
				ss_xx += (ss_temp*ss_temp);
				//-------------------------- STATS --------------------------
			}
		}
	}
	
	if(threadIdx.x==0){
		s_signal_mean = M/j;
		s_signal_sd   = sqrt(S/j);
	}
	
	__syncthreads();
	
	signal_mean = s_signal_mean;
	signal_sd   = s_signal_sd;
	//---- Calculation of the initial MSD
	//-----------------------------------------------------------
	
	//if(threadIdx.x==0) printf("Initial mean:%f; and standard deviation:%f;\n", signal_mean, signal_sd);
	
	
	//-------------------------- STATS --------------------------
	if(threadIdx.x==0){
		//printf("size:%f; alt_size:%f; ss_x:%f; ss_xx:%f; \n", (float) size, j/nElements, ss_x, ss_xx);
		d_stats[0]=signal_mean;
		d_stats[1]=signal_sd;
		d_stats[2]=ss_x/((float) (j/nElements)); //mean of sigma
		d_stats[3]=sqrt((ss_xx - (ss_x*ss_x)/((float) (j/nElements)))/((float) (j/nElements)));  //sigma of sigma
	}
	//-------------------------- STATS --------------------------
	
	
	
	//----------------------------------------------
	//---- Iterations with outlier rejection
	for(int f=0; f<nIterations; f++){
		pos=threadIdx.x;
		if(size>blockDim.x){
			M=0;
			S=0;
			j=0;
			
			//-------------------------- STATS --------------------------
			ss_x    = 0;
			ss_xx   = 0;
			//-------------------------- STATS --------------------------
			while (pos<size){
				Mt=__ldg(&d_input[3*pos]);
				if( (Mt/nElements > (signal_mean - multiplier*signal_sd)) && (Mt/nElements < (signal_mean + multiplier*signal_sd)) ){
					if(j==0){
						M = Mt;
						S = __ldg(&d_input[3*pos+1]);
						j = nElements;
						//-------------------------- STATS --------------------------
						ss_temp = sqrt(__ldg(&d_input[3*pos+1])/((float) nElements));
						ss_x    = ss_temp;
						ss_xx   = (ss_temp*ss_temp);
						//-------------------------- STATS --------------------------
					}
					else{
						jv=nElements;
						ftemp = ( jv/j*M - Mt);
						S = S + __ldg(&d_input[3*pos+1]) + (j/(jv*(j+jv)))*ftemp*ftemp;
						M = M + Mt;
						j=j+jv;
						//-------------------------- STATS --------------------------
						ss_temp = sqrt(__ldg(&d_input[3*pos+1])/((float) nElements));
						ss_x  += ss_temp;
						ss_xx += (ss_temp*ss_temp);
						//-------------------------- STATS --------------------------
					}
				}
				pos = pos + blockDim.x;
			}
			
			__syncthreads();
			
			Ms[threadIdx.x]=M;
			Ss[threadIdx.x]=S;
			js[threadIdx.x]=j;
			//-------------------------- STATS --------------------------
			s_ss_x[threadIdx.x]  = ss_x;
			s_ss_xx[threadIdx.x] = ss_xx;
			//-------------------------- STATS --------------------------
			// now all threads had saved their work, reduction follows		
			// first we must load initial values
			for(int i=(blockDim.x>>1); i>HALF_WARP; i=i>>1){
				if(threadIdx.x<i){
					jv=js[i + threadIdx.x];
					if(jv!=0){
						if(j==0){
							S = Ss[i + threadIdx.x];
							M = Ms[i + threadIdx.x];
							j = jv;
							//-------------------------- STATS --------------------------
							ss_x  = s_ss_x[threadIdx.x + i];
							ss_xx = s_ss_xx[threadIdx.x + i];
							//-------------------------- STATS --------------------------
						}
						else {
							ftemp = (jv/j*M - Ms[i + threadIdx.x]);
							S = S + Ss[i + threadIdx.x] + (j/(jv*(j+jv)))*ftemp*ftemp;
							M = M + Ms[i + threadIdx.x];
							j=j+jv;
							//-------------------------- STATS --------------------------
							ss_x  = ss_x + s_ss_x[threadIdx.x + i];
							ss_xx = ss_xx + s_ss_xx[threadIdx.x + i];
							//-------------------------- STATS --------------------------
						}
					}
					
					Ms[threadIdx.x]=M;
					Ss[threadIdx.x]=S;
					js[threadIdx.x]=j;
					//-------------------------- STATS --------------------------
					s_ss_x[threadIdx.x]  = ss_x;
					s_ss_xx[threadIdx.x] = ss_xx;
					//-------------------------- STATS --------------------------
				}
				__syncthreads();
			}
			
			// by now we should have only 32 partial results. shuffle reduction follows
			for(int q=HALF_WARP; q>0; q=q>>1){
				jv=__shfl_down(j, q);
				if(jv!=0){
					if(j==0) {
						S = __shfl_down(S, q);
						M = __shfl_down(M, q);
						j = jv;
						//-------------------------- STATS --------------------------
						ss_x  = __shfl_down(ss_x, q);
						ss_xx = __shfl_down(ss_xx, q);
						//-------------------------- STATS --------------------------
					}
					else {
						ftemp = (jv/j*M - __shfl_down(M, q));
						S = S + __shfl_down(S, q) + (j/(jv*(j+jv)))*ftemp*ftemp;
						M = M + __shfl_down(M, q);
						j=j+jv;	
						//-------------------------- STATS --------------------------
						ss_x  = ss_x + __shfl_down(ss_x, q);
						ss_xx = ss_xx + __shfl_down(ss_xx, q);
						//-------------------------- STATS --------------------------
					}

				}
			}
			
		}
		else {
			if(threadIdx.x==0){
				//printf("Doing one thread thing\n");
				M=0;
				S=0;
				j=0;
				for(pos=0; pos<size; pos++){
					Mt=__ldg(&d_input[3*pos]);
					if( (Mt/nElements > (signal_mean - multiplier*signal_sd)) && (Mt/nElements < (signal_mean + multiplier*signal_sd)) ){
						if(j==0){
							M=Mt;
							S=__ldg(&d_input[3*pos+1]);
							j=nElements;
							
							//-------------------------- STATS --------------------------
							ss_temp = sqrt(__ldg(&d_input[3*pos+1])/((float) nElements));
							ss_x    = ss_temp;
							ss_xx   = (ss_temp*ss_temp);
							//-------------------------- STATS --------------------------
						}
						else{
							jv=nElements;
							ftemp = ( jv/j*M - __ldg(&d_input[3*pos]) );
							S = S + __ldg(&d_input[3*pos+1]) + (j/(jv*(j+jv)))*ftemp*ftemp;
							M = M + __ldg(&d_input[3*pos]);
							j=j+jv;
							
							//-------------------------- STATS --------------------------
							ss_temp = sqrt(__ldg(&d_input[3*pos+1])/((float) nElements));
							ss_x  += ss_temp;
							ss_xx += (ss_temp*ss_temp);
							//-------------------------- STATS --------------------------
						}
					}
				}
			}
		}
		
		if(threadIdx.x==0){
			s_signal_mean = M/j;
			s_signal_sd   = sqrt(S/j);
		}
		
		__syncthreads();
		
		signal_mean = s_signal_mean;
		signal_sd   = s_signal_sd;
		
		
		
		//if(threadIdx.x==0) printf("Corrected mean:%f; and standard deviation:%f;\n", signal_mean, signal_sd);
		if(threadIdx.x==0){
			//printf("size:%f; alt_size:%f; ss_x:%f; ss_xx:%f; \n", (float) size, j/nElements, ss_x, ss_xx);
			d_stats[4*(f+1) + 0]=signal_mean;
			d_stats[4*(f+1) + 1]=signal_sd;
			d_stats[4*(f+1) + 2]=ss_x/((float) (j/nElements));
			d_stats[4*(f+1) + 3]=sqrt((ss_xx - (ss_x*ss_x)/((float) (j/nElements)))/((float) (j/nElements)));
		}
	}
	//---- Iterations with outlier rejection
	//----------------------------------------------
	
	
	
	//----------------------------------------------
	//---- Writing data
	if(threadIdx.x==0){
		d_output[0] = signal_mean;
		d_output[1] = signal_sd;
		d_output[2] = j;
	}
	//---- Writing data
	//----------------------------------------------
}

#endif
